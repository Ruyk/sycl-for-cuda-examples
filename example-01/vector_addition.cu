// Original source reproduced unmodified here from: 
// https://github.com/olcf/vector_addition_tutorials/blob/master/CUDA/vecAdd.cu


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *devA, double *devB, double *devC, int n) {
  // Get our global thread ID
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    devC[i] = devA[i] + devB[i];
}

int main(int argc, char *argv[]) {
  // Size of vectors
  constexpr const size_t N = 100000;
  // Size, in bytes, of each vector
  constexpr const size_t bytes = N * sizeof(double);


  // Device vectors
  double *devA, *devB, *devC;

  
  // Allocate memory for each vector on GPU
  hipMallocManaged(&devA, bytes);
  hipMallocManaged(&devB, bytes);
  hipMallocManaged(&devC, bytes);

  // Initialize vectors on host
  for (int i = 0; i < N; i++) {
    devA[i] = sin(i) * sin(i);
    devB[i] = cos(i) * cos(i);
  }

  // Number of threads in each thread block
  int blockSize = 1024;
  // Number of thread blocks in grid
  int gridSize = (int)ceil((float)N / blockSize);
  // Execute the kernel
  vecAdd<<<gridSize, blockSize>>>(devA, devB, devC, N);

  hipDeviceSynchronize();

  // Sum up vector c and print result divided by n, this should equal 1 within
  // error
  double sum = 0;
  for (int i = 0; i < N; i++)
    sum += devC[i];
  printf("final result: %f\n", sum / N);

  // Release device memory
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);

  return 0;
}
